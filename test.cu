#include <hip/hip_runtime.h>
#include <stdio.h>

void __global__ simplestream(double* a, double*b, int N)
{
  int tid= threadIdx.x+blockIdx.x*blockDim.x;
  for(int id=tid; id<N; id+=blockDim.x*gridDim.x)
  {
    a[id]=b[id]+id;
  }
}

int  main(int argc, char const *argv[])
{
  hipStream_t streamA, streamB;
  hipStreamCreate ( &streamA);
  hipStreamCreate ( &streamB);
  int N=1024*1024;
  int size=N*sizeof(double);
  double*ptr1;
  double*ptr2;
  double*ptr3;
  double*ptr4;
  double*h_ptr;
  hipHostAlloc(&h_ptr, N*sizeof(double),hipHostMallocDefault);
  dim3 threads = dim3(512, 1);
  dim3 blocks  = dim3(N / threads.x, 1);
  hipEvent_t start1, stop1;
  hipEvent_t start2, stop2;
  hipEventCreate(&start1);
  hipEventCreate(&stop1);
  hipEventCreate(&start2);
  hipEventCreate(&stop2);
  hipEventRecord(start1, 0);
  hipMallocAsync(&ptr1, size, streamA);
  hipMallocAsync(&ptr2, size, streamA);
  hipEventRecord(stop1, 0);
  simplestream<<<blocks, threads, 0, streamA>>>(ptr1,ptr2,N );
  hipMemcpy(h_ptr, ptr2, size, hipMemcpyHostToDevice);
  hipFreeAsync(ptr1,streamA);
  hipFreeAsync(ptr2,streamA);
  hipEventRecord(start2, 0);
  hipMallocAsync(&ptr3, size, streamB);
  hipMallocAsync(&ptr4, size, streamB);
  hipEventRecord(stop2, 0);
  simplestream<<<blocks, threads, 0, streamB>>>(ptr3,ptr4,N);
  hipMemcpy(h_ptr, ptr4, size, hipMemcpyHostToDevice);
  hipFreeAsync(ptr3,streamB); 
  hipFreeAsync(ptr4,streamB); 
  hipHostFree(h_ptr);
  float timer1,timer2;
  hipEventElapsedTime ( &timer1,  start1,  stop1 );
  hipEventElapsedTime ( &timer2,  start2,  stop2 );
  printf("%f,%f\n",timer1, timer2);
  hipEventDestroy(start1);
  hipEventDestroy(stop1);
  hipEventDestroy(start2);
  hipEventDestroy(stop2);
  hipStreamDestroy(streamB);
  hipStreamDestroy(streamA);
  return 0;

}

